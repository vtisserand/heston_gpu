#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "config.h"
#include "monte_carlo.h"

__global__ void initCurand(unsigned int seed, 
						   hiprandState* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void MC_Heston(hiprandState* state, 
                          float S0, 
                          float V0, 
                          float r, 
                          float kappa, 
                          float theta, 
                          float rho, 
                          float sigma, 
                          float dt, 
                          float K, 
                          int N, 
                          float *sum, 
                          int n) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState localState = state[idx];
    float2 Ws = make_float2(0.0f, 0.0f); // Initialize Ws

    float S = S0;
    float V = V0;

    // Dynamic allocation of shared memory
    extern __shared__ float A[];
    float* R1s = A;
    float* R2s = R1s + blockDim.x;

    for (int i = 0; i < N; i++) {
        Ws = hiprand_normal2(&localState);
        V += kappa * (theta - V) * dt * dt + sigma * sqrtf(fmaxf(0.0f, V)) * dt * Ws.x;
        S += r * S * dt * dt + sqrtf(V) * S * dt * (rho * Ws.x + sqrtf(1 - rho * rho) * Ws.y);
    }

    R1s[threadIdx.x] = expf(-r * dt * dt * N) * fmaxf(0.0f, S - K); // Call price
    R2s[threadIdx.x] = R1s[threadIdx.x] * R1s[threadIdx.x];

    __syncthreads(); // Block-level synchronisation

    int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            R1s[threadIdx.x] += R1s[threadIdx.x + i];
            R2s[threadIdx.x] += R2s[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Update global memory atomically
    if (threadIdx.x == 0) {
        atomicAdd(sum, R1s[0] / n);
        atomicAdd(sum + 1, R2s[0] / n);
    }
}


int main(void) {

	int NTPB = 1024;
	int NB = 1024;
	int n = NB * NTPB;
	float T = 5.0f;
	float S0 = 1.0f;
	float V0 = 0.1f;
	float K = 1.0;
	float sigma = 0.2f;
	float r = 0.0f;
    float kappa = 2.0f;
    float theta = 0.1f;
    float rho = -0.3f;
	int N = 1000;
	float dt = sqrtf(T/N);

	// Allow some memory for sum (options payoff at maturity) and random states.
	float *sum;
	hipMallocManaged(&sum, 2*sizeof(float)); // We save payoff and std
	hipMemset(sum, 0, 2*sizeof(float));

	hiprandState* states;
	hipMalloc(&states, n*sizeof(hiprandState));
	initCurand<<<NB, NTPB>>>(time(0), states);

	float Tim;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Launch one simulation on each thread.
	MC_Heston<<<NB, NTPB, 2*NTPB*sizeof(float)>>>(states, 
												  S0, 
												  V0, 
												  r, 
												  kappa, 
												  theta, 
												  rho, 
												  sigma, 
												  dt, 
												  K, 
												  N, 
												  sum, 
											 	  n);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&Tim, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("The estimated price is equal to %f\n", sum[0]);
	printf("error associated to a confidence interval of 95%% = %f\n",
		1.96 * sqrt((double)(sum[1] - (sum[0] * sum[0])))/sqrt((double)n));
	printf("Execution time %f ms\n", Tim);

	hipFree(sum);
	hipFree(states);

	return 0;
}